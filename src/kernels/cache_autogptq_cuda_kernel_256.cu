#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdint.h>

#if (defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 700) || defined(USE_ROCM)
// adapted from https://github.com/PanQiWei/AutoGPTQ/blob/main/autogptq_extension/cuda_256/autogptq_cuda_kernel_256.cu
__device__ __forceinline__ void atomicAdd(c10::Half* address, c10::Half val) {
    unsigned int *address_as_ui = reinterpret_cast<unsigned int *>(reinterpret_cast<char *>(address) - (reinterpret_cast<size_t>(address) & 2));
    unsigned int old = *address_as_ui;
    unsigned int assumed;

    do {
        assumed = old;
        unsigned short hsum = reinterpret_cast<size_t>(address) & 2 ? (old >> 16) : (old & 0xffff);
        hsum += val;
        old = reinterpret_cast<size_t>(address) & 2
                 ? (old & 0xffff) | (hsum << 16)
                 : (old & 0xffff0000) | hsum;
        old = atomicCAS(address_as_ui, assumed, old);

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
}
__device__ __forceinline__ void atomicAdd(__half* address, c10::Half val) {
    unsigned int * address_as_ui = (unsigned int *) ((char *)address - ((size_t)address & 2));
    unsigned int old = *address_as_ui;
    unsigned int assumed;

    do {
        assumed = old;
        __half_raw hsum;
        hsum.x = (size_t)address & 2 ? (old >> 16) : (old & 0xffff);
        half tmpres = __hadd(hsum, val);
        hsum = __half_raw(tmpres);
        old = (size_t)address & 2 ? (old & 0xffff) | (hsum.x << 16) : (old & 0xffff0000) | hsum.x;
        old = atomicCAS(address_as_ui, assumed, old);
    } while (assumed != old);
}
#endif

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const       int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width
);

template <typename scalar_t>
__global__ void VecQuant8BatchMatMulColumnCompressionKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
);

template <typename scalar_t>
__global__ void VecQuant4BatchMatMulColumnCompressionKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
);

template <typename scalar_t>
__global__ void VecQuant8BatchMatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
);

template <typename scalar_t>
__global__ void VecQuant4BatchMatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
);



template <typename scalar_t>
__global__ void VecQuant8BatchMatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
);

__global__ void VecQuant8BatchMatMulKernel_faster(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
);



__global__ void VecQuant8BatchMatMulKernel_faster_old(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width
);


template <typename scalar_t>
__global__ void VecQuant4BatchMatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
);


template <typename scalar_t>
__global__ void VecQuant8BatchMatMulColumnCompressionKernel_old(
    const  scalar_t* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
);

__global__ void VecQuant8BatchMatMulColumnCompressionKernel_faster(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
);

__global__ void VecQuant8BatchMatMulColumnCompressionKernel_faster_old(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
);


template <typename scalar_t>
__global__ void VecQuant4BatchMatMulColumnCompressionKernel_old(
    const  scalar_t* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
);


__global__ void VecQuant8BatchMatMulKernel_faster(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width
);


__global__ void VecQuant8BatchMatMulColumnCompressionKernel_faster(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
);

const int BLOCKWIDTH  = 128;
const int BLOCKHEIGHT8 =  32;
const int BLOCKHEIGHT4 =  16;
const int BLOCKHEIGHT_OLD4 =  128;
//const int BLOCKHEIGHT_OLD8 =  128;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

__device__ inline int as_int(int i) {
  return *reinterpret_cast<int*>(&i);
}

void vecquant8matmul_batched_column_compression_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int height = vec.size(3);
  int width = mat.size(3) * 4;

  dim3 blocks(
    (height + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_batched_cuda", ([&] {
      VecQuant8BatchMatMulColumnCompressionKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, heads, vec_row, height, width
      );
    })
  );

}

template <typename scalar_t>
__global__ void VecQuant8BatchMatMulColumnCompressionKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
) {
  int weight_total = batch * heads * height * width / 4;
  int input_total = batch * heads * vec_row * height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  // h is index of height with step being BLOCKWIDTH
  int h = BLOCKWIDTH * blockIdx.x;
  // w is index of width with step being 1
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= height) {
    return;
  }

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int k;
  scalar_t w_tmp;

  float weight[BLOCKWIDTH];

  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h + k < height; ++k){
        int i_w = (w / 4);
        int w_bit = (w % 4) * 8;

        int w_index = (batch_shift * height + h + k) * width / 4 + i_w;
        if (w_index >= weight_total || w >= width) {
          weight[k] = 0;
        } else {
          scalar_t scale = scales[batch_shift * height + h + k];
          scalar_t zero = zeros[batch_shift * height + h + k];
          w_tmp = ((as_unsigned(mat[w_index]) >> w_bit) & 0xFF);
          weight[k] = scale * (w_tmp - zero);
        }
      }

      scalar_t res;
      for (int vr = 0; vr < vec_row; ++vr){
          res = 0;
        int vec_index = (batch_shift * vec_row + vr) * height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = vec[vec_index];
        } else {
            blockvec[tid] = 0;
        }

        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h + k < height; ++k){
          // res is the dot product of BLOCKWIDTH elements (part of width)
            res += weight[k] * blockvec[k];
        }
        // add res to the final result, final matrix shape: (batch, vec_row, width)
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], res);
        }
        __syncthreads();
      }
    }
  }
}

void vecquant8matmul_batched_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int vec_height = vec.size(3);
  int height = mat.size(2);
  int width = mat.size(3);
  int zero_width = zeros.size(2);

  dim3 blocks(
    (height + BLOCKHEIGHT8 - 1) / BLOCKHEIGHT8,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_batched_cuda", ([&] {
      VecQuant8BatchMatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, heads, vec_row, vec_height, height, width, zero_width
      );
    })
  );

}

template <typename scalar_t>
__global__ void VecQuant8BatchMatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
) {
  int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * vec_height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  // h is index of height with step being BLOCKHEIGHT8
  int h = BLOCKHEIGHT8 * blockIdx.x;
  // w is index of width with step being 1
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= vec_height) {
    return;
  }

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  // i is index of mat of block first row
  int i = width * h + w;
  // if (i >= width * height) {
  //   return;
  // }
  int k;
  scalar_t w_tmp;

  int z_w = w / 4;
  int z_mod = (w % 4) * 8;

  float weight[BLOCKWIDTH];

  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h * 4 + k < vec_height; ++k){
        int k_w = (k / 4);
        int k_bit = (k % 4) * 8;

        int w_index = batch_shift * height * width + i + (k_w * width);
        if (w_index >= weight_total || w >= width) {
          weight[k] = 0;
        } else {
          scalar_t scale = scales[batch_shift * width + w];
          scalar_t zero;
          if (zero_width == width) {
            zero = zeros[batch_shift * width + w];
          } else {
            zero = scalar_t(((as_unsigned(zeros[batch_shift * zero_width + z_w]) >> z_mod) & 0xFF) + 1);
          }
          w_tmp = ((as_unsigned(mat[w_index]) >> k_bit) & 0xFF);
          weight[k] = scale * (w_tmp - zero);
        }
      }

      scalar_t res;
      for (int vr = 0; vr < vec_row; ++vr){
          res = 0;
        int vec_index = (batch_shift * vec_row + vr) * vec_height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = vec[vec_index];
        } else {
            blockvec[tid] = 0;
        }

        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h * 4 + k < vec_height; ++k){
          // res is the dot product of BLOCKWIDTH elements (part of width)
            res += weight[k] * blockvec[k];
        }
        // add res to the final result, final matrix shape: (batch, vec_row, width)
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], res);
        }
        __syncthreads();
      }
    }
  }
}


void vecquant8matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT8 - 1) / BLOCKHEIGHT8,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_cuda", ([&] {
      VecQuant8MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(),
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const       int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width
) {
  int h = BLOCKHEIGHT8 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = h * 4;
  int k;
  unsigned int g;
  scalar_t w_tmp;

  int z_w = w / 4;
  int z_mod = (w % 4) * 8;

  float weight[BLOCKWIDTH];

  for (k = 0; k <  BLOCKWIDTH; ++k){
    int k_w = (k / 4);
    int k_bit = (k % 4) * 8;

      g = as_int(g_idx[g_h + k]);
      scalar_t scale = scales[g * width + w];
      scalar_t zero = scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xFF) + 1);

      w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0xFF);

    weight[k] = scale * (w_tmp - zero);
  }


  scalar_t res;
  for (int b = 0; b < batch; ++b){
      res = 0;
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
    for (k = 0; k <  BLOCKWIDTH; ++k){
      res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
    __syncthreads();
  }
}



void vecquant4matmul_batched_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int vec_height = vec.size(3);
  int height = mat.size(2);
  int width = mat.size(3);
  int zero_width = zeros.size(2);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_batched_cuda", ([&] {
      VecQuant4BatchMatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, heads, vec_row, vec_height, height, width, zero_width
      );
    })
  );

}

template <typename scalar_t>
__global__ void VecQuant4BatchMatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
) {
  int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * vec_height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  // h is index of height with step being BLOCKHEIGHT4
  int h = BLOCKHEIGHT4 * blockIdx.x;
  // w is index of width with step being 1
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= vec_height) {
    return;
  }

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  // i is index of mat of block first row
  int i = width * h + w;
  int k;
  scalar_t w_tmp;

  int z_w = w / 8;
  int z_mod = (w % 8) * 4;

  float weight[BLOCKWIDTH];

  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h * 8 + k < vec_height; ++k){
        int k_w = (k / 8);
        int k_bit = (k % 8) * 4;

        int w_index = batch_shift * height * width + i + (k_w * width);
        if (w_index >= weight_total || w >= width) {
          weight[k] = 0;
        } else {
          scalar_t scale = scales[batch_shift * width + w];
          scalar_t zero;
          if (zero_width == width) {
            zero = zeros[batch_shift * width + w];
          } else {
            zero = scalar_t(((as_unsigned(zeros[batch_shift * zero_width + z_w]) >> z_mod) & 0xF));
          }
          w_tmp = ((as_unsigned(mat[w_index]) >> k_bit) & 0xF);
          weight[k] = scale * (w_tmp - zero);
        }
      }

      scalar_t res;
      for (int vr = 0; vr < vec_row; ++vr){
          res = 0;
        int vec_index = (batch_shift * vec_row + vr) * vec_height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = vec[vec_index];
        } else {
            blockvec[tid] = 0;
        }

        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h * 8 + k < vec_height; ++k){
          // res is the dot product of BLOCKWIDTH elements (part of width)
            res += weight[k] * blockvec[k];
        }
        // add res to the final result, final matrix shape: (batch, vec_row, width)
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], res);
        }
        __syncthreads();
      }
    }
  }
}



void vecquant4matmul_batched_column_compression_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int height = vec.size(3);
  int width = mat.size(3) * 8;

  dim3 blocks(
    (height + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_batched_cuda", ([&] {
      VecQuant4BatchMatMulColumnCompressionKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, heads, vec_row, height, width
      );
    })
  );

}

template <typename scalar_t>
__global__ void VecQuant4BatchMatMulColumnCompressionKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
) {
  int weight_total = batch * heads * height * width / 8;
  int input_total = batch * heads * vec_row * height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  // h is index of height with step being BLOCKWIDTH
  int h = BLOCKWIDTH * blockIdx.x;
  // w is index of width with step being 1
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= height) {
    return;
  }

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int k;
  scalar_t w_tmp;

  float weight[BLOCKWIDTH];

  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h + k < height; ++k){
        int i_w = (w / 8);
        int w_bit = (w % 8) * 4;

        int w_index = (batch_shift * height + h + k) * width / 8 + i_w;
        if (w_index >= weight_total || w >= width) {
          weight[k] = 0;
        } else {
          scalar_t scale = scales[batch_shift * height + h + k];
          scalar_t zero = zeros[batch_shift * height + h + k];
          w_tmp = ((as_unsigned(mat[w_index]) >> w_bit) & 0xF);
          weight[k] = scale * (w_tmp - zero);
        }
      }

      scalar_t res;
      for (int vr = 0; vr < vec_row; ++vr){
          res = 0;
        int vec_index = (batch_shift * vec_row + vr) * height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = vec[vec_index];
        } else {
            blockvec[tid] = 0;
        }

        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h + k < height; ++k){
          // res is the dot product of BLOCKWIDTH elements (part of width)
            res += weight[k] * blockvec[k];
        }
        // add res to the final result, final matrix shape: (batch, vec_row, width)
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], res);
        }
        __syncthreads();
      }
    }
  }
}


void vecquant8matmul_batched_old_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int vec_height = vec.size(3);
  int height = mat.size(2);
  int width = mat.size(3);
  int zero_width = zeros.size(2);

  dim3 blocks(
    (height + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_batched_old_cuda", ([&] {
      VecQuant8BatchMatMulKernel_old<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<uint8_t>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, heads, vec_row, vec_height, height, width, zero_width
      );
    })
  );
}


template <typename scalar_t>
__global__ void VecQuant8BatchMatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
) {
  int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * vec_height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  // h is index of height with step being BLOCKHEIGHT8
  int h = BLOCKWIDTH * blockIdx.x;
  // w is index of width with step being 1
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= vec_height) {
    return;
  }

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  // i is index of mat of block first row
  int i = width * h + w;
  int k;
  scalar_t w_tmp;

  float weight[BLOCKWIDTH];
  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h + k < vec_height; ++k){
        int k_w = k;
        int w_index = batch_shift * height * width + i + (k_w * width);
        if (w_index >= weight_total || w >= width) {
          weight[k] = 0;
        } else {
          scalar_t scale = scales[batch_shift * width + w];
          scalar_t zero = zeros[batch_shift * width + w];
          w_tmp = as_unsigned(mat[w_index]);
          weight[k] = scale * (w_tmp - zero);
        }
      }

      scalar_t res;
      for (int vr = 0; vr < vec_row; ++vr){
          res = 0;
        int vec_index = (batch_shift * vec_row + vr) * vec_height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = vec[vec_index];
        } else {
            blockvec[tid] = 0;
        }

        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h + k < vec_height; ++k){
          // res is the dot product of BLOCKWIDTH elements (part of width)
            res += weight[k] * blockvec[k];
        }
        // add res to the final result, final matrix shape: (batch, vec_row, width)
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], res);
        }
        __syncthreads();
      }
    }
  }
}



void vecquant8matmul_batched_faster_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int vec_height = vec.size(3);
  int height = mat.size(2);
  int width = mat.size(3);
  int zero_width = zeros.size(2);

  dim3 blocks(
    (height + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant8BatchMatMulKernel_faster<<<blocks, threads>>>(
    (half*) vec.data_ptr(),
    (uint8_t*) mat.data_ptr(),
    (half*) mul.data_ptr(),
    (half*) scales.data_ptr(),
    (half*) zeros.data_ptr(),
    batch, heads, vec_row, vec_height, height, width, zero_width
  );
}



__global__ void VecQuant8BatchMatMulKernel_faster(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
) {
  //int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * vec_height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  int h = BLOCKWIDTH * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= height) {
    return;
  }

  __shared__ float blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int k;
  float w_tmp;

  float weight[BLOCKWIDTH];
  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h + k < vec_height; ++k){
        int k_w = k;
        int w_index = batch_shift * height * width + i + (k_w * width);
        float scale = __half2float(scales[batch_shift * width + w]);
        float zero = __half2float(zeros[batch_shift * width + w]);
        w_tmp = as_unsigned(mat[w_index]);
        weight[k] = scale *(w_tmp-zero);
      }

      float res;
      for (int vr = 0; vr < vec_row; ++vr){
        res = 0;
        int vec_index = (batch_shift * vec_row + vr) * vec_height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = __half2float(vec[vec_index]);
        } else {
            blockvec[tid] = 0;
        }
        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h + k < vec_height; ++k){
            float temp_res = weight[k]*blockvec[k];
            res += temp_res;
        }
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], __float2half(res));
        }
        __syncthreads();
      }
    }
  }
}




void vecquant8matmul_batched_column_compression_faster_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int height = vec.size(3);
  int width = mat.size(3);

  dim3 blocks(
    (height + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant8BatchMatMulColumnCompressionKernel_faster<<<blocks, threads>>>(
    (half*) vec.data_ptr(),
    (uint8_t*) mat.data_ptr(),
    (half*) mul.data_ptr(),
    (half*) scales.data_ptr(),
    (half*) zeros.data_ptr(),
    batch, heads, vec_row, height, width
  );

}

__global__ void VecQuant8BatchMatMulColumnCompressionKernel_faster(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
) {
  //int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  int h = BLOCKWIDTH * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= height) {
    return;
  }

  __shared__ float blockvec[BLOCKWIDTH];
  int k;
  float w_tmp;
  float weight[BLOCKWIDTH];

  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH; ++k){
        int w_index = (batch_shift * height + h + k) * width  + w;
        float scale = __half2float(scales[batch_shift * height + h + k]);
        float zero = __half2float(zeros[batch_shift * height + h + k]);
        w_tmp = mat[w_index];
        weight[k] = scale * (w_tmp-zero);
      }

      float res;
      for (int vr = 0; vr < vec_row; ++vr){
        res = 0;
        int vec_index = (batch_shift * vec_row + vr) * height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = __half2float(vec[vec_index]);
        } else {
            blockvec[tid] = 0;
        }
        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH; ++k){
            res += weight[k]*blockvec[k];
        }
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], __float2half(res));
        }
        __syncthreads();
      }
    }
  }
}



void vecquant8matmul_batched_column_compression_old_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int height = vec.size(3);
  int width = mat.size(3);

  dim3 blocks(
    (height + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_batched_column_compression_old_cuda", ([&] {
      VecQuant8BatchMatMulColumnCompressionKernel_old<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<uint8_t>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, heads, vec_row, height, width
      );
    })
  );

}

template <typename scalar_t>
__global__ void VecQuant8BatchMatMulColumnCompressionKernel_old(
    const  scalar_t* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
) {
  int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  // h is index of height with step being BLOCKWIDTH
  int h = BLOCKWIDTH * blockIdx.x;
  // w is index of width with step being 1
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= height) {
    return;
  }

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int k;
  scalar_t w_tmp;

  float weight[BLOCKWIDTH];

  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h + k < height; ++k){
        int w_index = (batch_shift * height + h + k) * width  + w;
        if (w_index >= weight_total || w >= width) {
          weight[k] = 0;
        } else {
          scalar_t scale = scales[batch_shift * height + h + k];
          scalar_t zero = zeros[batch_shift * height + h + k];
          w_tmp = mat[w_index];
          weight[k] = scale * (w_tmp - zero);
        }
      }

      scalar_t res;
      for (int vr = 0; vr < vec_row; ++vr){
          res = 0;
        int vec_index = (batch_shift * vec_row + vr) * height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = vec[vec_index];
        } else {
            blockvec[tid] = 0;
        }

        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h + k < height; ++k){
          // res is the dot product of BLOCKWIDTH elements (part of width)
            res += weight[k] * blockvec[k];
        }
        // add res to the final result, final matrix shape: (batch, vec_row, width)
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], res);
        }
        __syncthreads();
      }
    }
  }
}


void vecquant4matmul_batched_old_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int vec_height = vec.size(3);
  int height = mat.size(2);
  int width = mat.size(3);
  int zero_width = zeros.size(2);

  dim3 blocks(
    (height + BLOCKHEIGHT_OLD4 - 1) / BLOCKHEIGHT_OLD4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_batched_old_cuda", ([&] {
      VecQuant4BatchMatMulKernel_old<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<uint8_t>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, heads, vec_row, vec_height, height, width, zero_width
      );
    })
  );

}

template <typename scalar_t>
__global__ void VecQuant4BatchMatMulKernel_old(
    const  scalar_t* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width,
    int zero_width
) {
  int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * vec_height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  // h is index of height with step being BLOCKHEIGHT_OLD4
  int h = BLOCKHEIGHT_OLD4 * blockIdx.x;
  // w is index of width with step being 1
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= vec_height) {
    return;
  }

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  // i is index of mat of block first row
  int i = width * h + w;
  int k;
  scalar_t w_tmp;

  float weight[BLOCKWIDTH];
  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h*2 + k < vec_height; ++k){
        int k_w = (k / 2);
        int k_bit = (k % 2) * 4;
        int w_index = batch_shift * height * width + i + (k_w * width);
        if (w_index >= weight_total || w >= width) {
          weight[k] = 0;
        } else {
          scalar_t scale = scales[batch_shift * width + w];
          scalar_t zero = zeros[batch_shift * width + w];
          w_tmp = ((as_unsigned(mat[w_index]) >> k_bit) & 0xF);
          weight[k] = scale * (w_tmp - zero);
        }
      }

      scalar_t res;
      for (int vr = 0; vr < vec_row; ++vr){
          res = 0;
        int vec_index = (batch_shift * vec_row + vr) * vec_height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = vec[vec_index];
        } else {
            blockvec[tid] = 0;
        }

        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h*2 + k < vec_height; ++k){
          // res is the dot product of BLOCKWIDTH elements (part of width)
            res += weight[k] * blockvec[k];
        }
        // add res to the final result, final matrix shape: (batch, vec_row, width)
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], res);
        }
        __syncthreads();
      }
    }
  }
}





void vecquant4matmul_batched_column_compression_old_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int height = vec.size(3);
  int width = mat.size(3);

  dim3 blocks(
    (height + BLOCKHEIGHT_OLD4 - 1) / BLOCKHEIGHT_OLD4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_batched_column_compression_old_cuda", ([&] {
      VecQuant4BatchMatMulColumnCompressionKernel_old<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<uint8_t>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, heads, vec_row, height, width
      );
    })
  );

}

template <typename scalar_t>
__global__ void VecQuant4BatchMatMulColumnCompressionKernel_old(
    const  scalar_t* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int height,
    int width
) {
  int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  // h is index of height with step being BLOCKWIDTH
  int h = BLOCKHEIGHT_OLD4 * blockIdx.x;
  // w is index of width with step being 1
  int w = BLOCKWIDTH * blockIdx.y + tid;
  if (w >= width && tid >= height) {
    return;
  }

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int k;
  scalar_t w_tmp;

  float weight[BLOCKWIDTH];

  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      for (k = 0; k <  BLOCKWIDTH && h*2 + k < height; ++k){
        int k_w = (k / 2);
        int k_bit = (k % 2) * 4;
        int w_index = (batch_shift * height + h + k) * width  + k_w;
        if (w_index >= weight_total || w >= width) {
          weight[k] = 0;
        } else {
          scalar_t scale = scales[batch_shift * height + h + k];
          scalar_t zero = zeros[batch_shift * height + h + k];
          w_tmp = ((as_unsigned(mat[w_index]) >> k_bit) & 0xF);
          weight[k] = scale * (w_tmp - zero);
        }
      }

      scalar_t res;
      for (int vr = 0; vr < vec_row; ++vr){
          res = 0;
        int vec_index = (batch_shift * vec_row + vr) * height + blockIdx.x * BLOCKWIDTH + tid;
        if (vec_index < input_total) {
            blockvec[tid] = vec[vec_index];
        } else {
            blockvec[tid] = 0;
        }

        __syncthreads();
          for (k = 0; k <  BLOCKWIDTH && h*2 + k < height; ++k){
          // res is the dot product of BLOCKWIDTH elements (part of width)
            res += weight[k] * blockvec[k];
        }
        // add res to the final result, final matrix shape: (batch, vec_row, width)
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (out_index < out_total) {
            atomicAdd(&mul[out_index], res);
        }
        __syncthreads();
      }
    }
  }
}





void vecquant8matmul_batched_faster_old_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2);
  int vec_height = vec.size(3);
  int height = mat.size(2);
  int width = mat.size(3);

  dim3 blocks(
    (height + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant8BatchMatMulKernel_faster_old<<<blocks, threads>>>(
    (half*) vec.data_ptr(),
    (uint8_t*) mat.data_ptr(),
    (half*) mul.data_ptr(),
    (half*) scales.data_ptr(),
    (half*) zeros.data_ptr(),
    batch, heads, vec_row, vec_height, height, width
  );
}


__global__ void VecQuant8BatchMatMulKernel_faster_old(
    const  half* __restrict__ vec,
    const  uint8_t* __restrict__ mat,
           half* __restrict__ mul,
    const  half* __restrict__ scales,
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row,
    int vec_height,
    int height,
    int width
) {
 int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * vec_height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  const int BLOCKWIDTH_half = BLOCKWIDTH/2;

  int h = BLOCKWIDTH * blockIdx.x; //head_dim, dim=-1
  int w = BLOCKWIDTH * blockIdx.y + tid; //seq-len, +0-256 ,dim=-2
  /*
  if (w >= width && tid >= vec_height) {
    return;
  }
  */
  __shared__ half blockvec[BLOCKWIDTH]; //256
  int i = width * h + w;
  int k;

  half w_tmp1 = __float2half(0);
  half w_tmp2 = __float2half(0);

  half2 weight[BLOCKWIDTH_half];
  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      //int zero_index = batch_shift;
      for (k = 0; k <  BLOCKWIDTH_half; ++k){
        int w_index1 = batch_shift * height * width + i + (2 * k * width); // [batch,head,h+k, w]
        int w_index2 = batch_shift * height * width + i + ((2 * k + 1) * width);
        int zero_index = batch_shift * width + w; // [batch,head, w]
        if (w_index1 >= weight_total || w >= width || (2 * k + h) >= height) {
          weight[k] = __float2half2_rn(0);
        } else {
            float zero_f=__half2float(zeros[zero_index]);
            float scale_f= __half2float(scales[zero_index]);
            if (w_index2 >= weight_total){
              w_tmp1 = __float2half((as_unsigned(mat[w_index1]) -zero_f)*scale_f);
              w_tmp2 = __float2half(0);
              weight[k] = __halves2half2(w_tmp1,w_tmp2);
              //printf("zero_index is %d w is %d height is %d width is %d w_index1 is %d w_tmp1 is %f w_tmp2 is %f zero is %f scale is %f low is %f high is %f \n ",zero_index,w,height, width,w_index1,__half2float(w_tmp1),__half2float(w_tmp2),zero_f,scale_f,__low2float(weight[k]),__high2float(weight[k]));
            }else{
              w_tmp1 = __int2half_rn(as_unsigned(mat[w_index1]));
              w_tmp2 = __int2half_rn(as_unsigned(mat[w_index2]));

              //weight[k] = __hmul2(__hsub2(__halves2half2(w_tmp1,w_tmp2), __halves2half2(zero,zero)),__halves2half2(scale,scale));
              weight[k] = __hfma2(__halves2half2(w_tmp1,w_tmp2), __float2half2_rn(scale_f), __float2half2_rn(-(scale_f * zero_f)));
              //printf("zero_index1 is %d zero_index2 is %d k is %d head is %d w is %d h is %d height is %d width is %d w_index1 is %d w_index2 is %d zero is %f scale is %f low is %f high is %f \n ",zero_index1,zero_index2,k,head,w,h,height, width,w_index1,w_index2,__half2float(zero1),__half2float(scale1),__low2float(weight[k]),__high2float(weight[k]));
            }
        }
      }


      for (int vr = 0; vr < vec_row; ++vr){
        float res=0;
        int vec_index = (batch_shift * vec_row + vr) * height + blockIdx.x * BLOCKWIDTH + tid;
        int out_index = (batch_shift * vec_row + vr) * width + w;
        if (vec_index < input_total) {
            //blockvec[tid] = __half2float(vec[vec_index]);// [batch, head, vr, tid(seq_len dim+)]
            blockvec[tid] = vec[vec_index];
            //printf("width is %d height is %d h is %d w is %d vec_index is %d out_index is %d vec_row is %d vec_height is %d,vr is %d tid is %d blockvec is %f\n",width,height, h,w,vec_index,out_index,vec_row,vec_height,vr,tid,blockvec[tid]);
        } else {
            blockvec[tid] = __float2half(0);
        }
        __syncthreads();
        if (out_index < out_total) {
          for (k = 0; k <  BLOCKWIDTH_half; ++k){
            half2 res2 = __hmul2(weight[k],__halves2half2(blockvec[2*k],blockvec[2*k+1]));
            res += __low2float(res2) + __high2float(res2);
          }
          atomicAdd(&mul[out_index], __float2half(res));
        }
        __syncthreads();
      }
    }
  }
}


void vecquant8matmul_batched_column_compression_faster_old_cuda(
  torch::Tensor vec,  // [batch,heads, seq_q, seq_v]
  torch::Tensor mat, // [batch,heads, seq_v, head_dim]
  torch::Tensor mul,  // [batch,heads, seq_q,head_dim]
  torch::Tensor scales, // [batch,heads, head_dim]
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int heads = vec.size(1);
  int vec_row = vec.size(2); //ql
  int height = mat.size(2); //vl
  int width = mat.size(3); //head_dim

  dim3 blocks(
    (height + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant8BatchMatMulColumnCompressionKernel_faster_old<<<blocks, threads>>>(
    (half*) vec.data_ptr(),
    (uint8_t*) mat.data_ptr(),
    (half*) mul.data_ptr(),
    (half*) scales.data_ptr(),
    (half*) zeros.data_ptr(),
    batch, heads, vec_row, height, width
  );

}


__global__ void VecQuant8BatchMatMulColumnCompressionKernel_faster_old(
    const  half* __restrict__ vec,  // [batch,heads, seq_q, seq_v]
    const  uint8_t* __restrict__ mat, // [batch,heads, seq_v, head_dim]
           half* __restrict__ mul, // [batch,heads, seq_q,head_dim]
    const  half* __restrict__ scales, // [batch,heads, seq_v]
    const  half* __restrict__ zeros,
    int batch,
    int heads,
    int vec_row, //seq_q
    int height, //seq_v
    int width //head_dim
) {
  int weight_total = batch * heads * height * width;
  int input_total = batch * heads * vec_row * height;
  int out_total = batch * heads * vec_row * width;
  int tid = threadIdx.x;
  int h = BLOCKWIDTH * blockIdx.x; // vl
  int w = BLOCKWIDTH * blockIdx.y + tid; //head_dim + block
  if (w >= width && tid >= height) {
    return;
  }
  __shared__ half blockvec[BLOCKWIDTH];
  int k;
  half w_tmp1 = __float2half(0);
  half w_tmp2 = __float2half(0);
  int i = width * h + w;
  const int BLOCKWIDTH_half = BLOCKWIDTH/2;
  half2 weight[BLOCKWIDTH_half];

  for (int b = 0; b < batch; ++b){
    for (int head = 0; head < heads; ++head){
      int batch_shift = b * heads + head;
      //int zero_index = batch_shift;
      for (k = 0; k <  BLOCKWIDTH_half; ++k){
        int w_index1 = batch_shift * height * width + i + (2 * k) * width; // [batch,head, h+k, w]
        int w_index2 = batch_shift * height * width + i + ((2 * k + 1) * width);
        int zero_index1 = batch_shift * height + h + 2*k; // [batch,head, w]
        int zero_index2 = batch_shift * height + h + 2*k+1; // [batch,head, w]

        if (w_index1 >= weight_total || (2 * k + h)>=height) {
          weight[k]=__float2half2_rn(0);
        } else{
            //int zero_index = batch_shift + h; // [batch,head, w]
            //float scale_f1 = __half2float(scales[zero_index1]);
            //float zero_f1 =  __half2float(zeros[zero_index1]);
            if (w_index2>=weight_total){
              w_tmp1 = __float2half((as_unsigned(mat[w_index1]) - __half2float(zeros[zero_index1]))* __half2float(scales[zero_index1]));
              w_tmp2 = __float2half(0);
              weight[k] = __halves2half2(w_tmp1,w_tmp2);
              //printf("zero_index is %d k is %d w is %d head is %d height is %d width is %d w_index1 is %d w_tmp1 is %f w_tmp2 is %f zero is %f scale is %f low is %f high is %f \n ",zero_index,k,w,head,height, width,w_index1,__half2float(w_tmp1),__half2float(w_tmp2),zero_f,scale_f,__low2float(weight[k]),__high2float(weight[k]));
            }else{
              w_tmp1 = __int2half_rn(as_unsigned(mat[w_index1]));
              w_tmp2 = __int2half_rn(as_unsigned(mat[w_index2]));
              half zero1=zeros[zero_index1];
              half zero2=zeros[zero_index2];
              half scale1=scales[zero_index1];
              half scale2=scales[zero_index2];
              weight[k] = __hmul2(__hsub2(__halves2half2(w_tmp1,w_tmp2), __halves2half2(zero1,zero2)),__halves2half2(scale1,scale2));
              //weight[k] = __hfma2(__halves2half2(w_tmp1,w_tmp2), __float2half2_rn(scale_f), __float2half2_rn(-(scale_f * zero_f)));
              //printf("zero_index1 is %d zero_index2 is %d k is %d head is %d w is %d h is %d height is %d width is %d w_index1 is %d w_index2 is %d zero is %f scale is %f low is %f high is %f \n ",zero_index1,zero_index2,k,head,w,h,height, width,w_index1,w_index2,__half2float(zero1),__half2float(scale1),__low2float(weight[k]),__high2float(weight[k]));
            }
          }
       }


      for (int vr = 0; vr < vec_row; ++vr){
        float res=0;
        int vec_index = (batch_shift * vec_row + vr) * height + blockIdx.x * BLOCKWIDTH + tid;
        int out_index = (batch_shift * vec_row + vr) * width + w;

        if (vec_index < input_total) {
            //blockvec[tid] = __half2float(vec[vec_index]);
            blockvec[tid] = vec[vec_index];
            //printf("vec_index is %d out_index is %d vec_row is %d ,vr is %d tid is %d blockvec is %f\n",vec_index,out_index,vec_row,vr,tid,blockvec[tid]);
        } else {
            blockvec[tid] = __float2half(0);
            //blockvec[tid] = 0;
        }
        __syncthreads();
        if (out_index < out_total) {
            for (k = 0; k <  BLOCKWIDTH_half; ++k){
                half2 res2 = __hmul2(weight[k],__halves2half2(blockvec[2*k],blockvec[2*k+1]));
                res += __low2float(res2) + __high2float(res2);
            }
            atomicAdd(&mul[out_index], __float2half(res));
        }
        __syncthreads();
      }
    }
  }
}
